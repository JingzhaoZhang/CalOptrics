#include "hip/hip_runtime.h"
#include <thrust/version.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <cusp/version.h>
#include <cusp/complex.h>

#include <iostream>
#include <list>
#include <vector>

#include "C:\Users\Diiv\git\CalOptrics\Cpp_library_windows\CalOptrics\CalOptrics\caloptrics.h"

void thrustVersionAndVectorExamples(); 
void thrustCopyFillSequenceExamples();
void thrustVectorListExample();
void anotherThrustTest();
void quitProgramPrompt(bool);
void coDataTypeSanityChecks();
void thrustMULTIPLYExample();

using namespace co;

int main(void)
{
    //thrustVersionAndVectorExamples();
	//thrustCopyFillSequenceExamples();
	//thrustVectorListExample();
	//anotherThrustTest();
	//coDataTypeSanityChecks();
	//thrustMULTIPLYExample();
	
	CudaArray<int> nums1 = CudaArray<int>(1, 10, 1);
	CudaArray<cfloat> nums2 = CudaArray<cfloat>(1, 3, cfloat(1,1));
	CudaArray<int> nums3 = CudaArray<int>(1, 10, 2);
	CudaArray<cfloat> nums4 = CudaArray<cfloat>(1, 10, cfloat(2,3));
	
	
	std::cout << nums1.dims() << std::endl;
	std::cout << nums1.elements() << std::endl;
	std::cout << nums1.isRowVector() << std::endl;
	std::cout << nums1.isColumnVector() << std::endl;
	std::cout << nums1.isScalar() << std::endl;


	std::cout << std::endl;

	CudaArray<int> nums5 = CudaArray<int>(1, 10, 0);
	plus<int>(nums5, nums1, nums3);

	CudaArray<bool> bool1 = CudaArray<bool>(10, 1, false);
	not<bool>(bool1);

	print_matrix("nums5", nums5);
	print_matrix("boo1", bool1);


	quitProgramPrompt(true);
    return 0;
}

/*
void coDataTypeSanityChecks()
{
	Float f1 = Float(3.14f);
	Float f2 = Float(2.14f);
	std::cout << (f1.val()==3.14f) << std::endl;
	std::cout << ((f1*f1-3.14f*3.14f).val() < .0001) << std::endl;
	std::cout << (((f1+f2)-(3.14f+2.14f)).val() < .0001) << std::endl;

	Double d1 = Double(3.14);
	Double d2 = Double(2.14);
	std::cout << (d1.val()==3.14) << std::endl;
	std::cout << (((d1*d1-3.14*3.14)).val() < .0001) << std::endl;
	std::cout << (((d1+d2)-(3.14+2.14)).val() < .0001) << std::endl;
	

	Bool b1 = Bool(true);
	Bool b2 = Bool(false);
	std::cout << (b1.val() == true) << std::endl;
	std::cout << (b2.val() == false) << std::endl;
	std::cout << (b1 != b2) << std::endl;
	std::cout << (!(b1 == b2)) << std::endl;
	std::cout << (!(b1 && b2)).val() << std::endl;

	Int i1 = Int(4);
	Int i2 = Int(9);
	std::cout << (i1.val()==4) << std::endl;
	std::cout << ((i1*i1).val() == 16) << std::endl;
	std::cout << ((i1+i2).val() == 13) << std::endl;

	hipfftComplex c1;
	c1.x = 2;
	c1.y = 3;
	CFloat cf1 = CFloat(c1);

	hipfftComplex c2;
	c2.x = 2;
	c2.y = 2;
	CFloat cf2 = CFloat(c2);

	std::cout << cf1.val().x << " " << cf1.val().y << std::endl;
	std::cout << cf2.val().x << " " << cf2.val().y << std::endl;
	std::cout << norm(cf1) << std::endl;
	std::cout << abs(cf1) << std::endl;
	std::cout << conj(cf2).val().x << " " << conj(cf2).val().y << std::endl;
	std::cout << (cf1+cf2).val().x << " " << (cf1+cf2).val().y << std::endl;
	std::cout << (cf1*cf2).val().x << " " << (cf1*cf2).val().y << std::endl;
	std::cout << (cf2/cf2).val().x << " " << (cf2/cf2).val().y << std::endl;

	hipfftDoubleComplex c3;
	c3.x = 2;
	c3.y = 3;
	CDouble cd1 = CDouble(c3);

	hipfftDoubleComplex c4;
	c4.x = 2;
	c4.y = 2;
	CDouble cd2 = CDouble(c4);

	std::cout << cd1.val().x << " " << cd1.val().y << std::endl;
	std::cout << cd2.val().x << " " << cd2.val().y << std::endl;
	std::cout << norm(cd1) << std::endl;
	std::cout << abs(cd1) << std::endl;
	std::cout << conj(cd2).val().x << " " << conj(cd2).val().y << std::endl;
	std::cout << (cd1+cd2).val().x << " " << (cd1+cd2).val().y << std::endl;
	std::cout << (cd1*cd2).val().x << " " << (cd1*cd2).val().y << std::endl;
	std::cout << (cd2/cd2).val().x << " " << (cd2/cd2).val().y << std::endl;
}
*/

void anotherThrustTest()
{
	thrust::device_vector<int>* D;
	thrust::host_vector<int>* H;
	D = new thrust::device_vector<int>(3,1);
	H = new thrust::host_vector<int>(*D);

	std::vector<int>* nums = new std::vector<int>;
	nums->push_back(1);
	nums->push_back(1);
	nums->push_back(1);
	thrust::host_vector<int>* N = new thrust::host_vector<int>(*nums);


	for(int i = 0;i < N->size(); i++){
		std::cout << "N[" << i << "]= " << (*N)[i] << std::endl; 
	}
}

void thrustVectorListExample()
{
	// create an STL list with 4 values
	std::list<int> stl_list;

	stl_list.push_back(10);
	stl_list.push_back(20);
	stl_list.push_back(30);
	stl_list.push_back(40);

	// initialize a device_vector with the list
	thrust::device_vector<int> D(stl_list.begin(), stl_list.end());

	// print D
    for(int i = 0; i < D.size(); i++)
        std::cout << "D[" << i << "] = " << D[i] << std::endl;

	std::cout << std::endl;

	// copy a device_vector into an STL vector
	std::vector<int> stl_vector(D.size());
	thrust::copy(D.begin(), D.end(), stl_vector.begin());

	// print stl_vector
    for(int i = 0; i < stl_vector.size(); i++)
        std::cout << "stl_vector[" << i << "] = " << stl_vector[i] << std::endl;
}

void thrustCopyFillSequenceExamples()
{
	 // initialize all ten integers of a device_vector to 1
    thrust::device_vector<int> D(10, 1);

    // set the first seven elements of a vector to 9
    thrust::fill(D.begin(), D.begin() + 7, 9);

    // initialize a host_vector with the first five elements of D
    thrust::host_vector<int> H(D.begin(), D.begin() + 5);

    // set the elements of H to 0, 1, 2, 3, ...
    thrust::sequence(H.begin(), H.end());

    // copy all of H back to the beginning of D
    thrust::copy(H.begin(), H.end(), D.begin());

    // print D
    for(int i = 0; i < D.size(); i++)
        std::cout << "D[" << i << "] = " << D[i] << std::endl;
}


void thrustVersionAndVectorExamples()
{
	int major = THRUST_MAJOR_VERSION;
    int minor = THRUST_MINOR_VERSION;
	int cusp_major = CUSP_MAJOR_VERSION;
    int cusp_minor = CUSP_MINOR_VERSION;

    std::cout << "Thrust v" << major << "." << minor << std::endl;
	std::cout << "Cusp v" << cusp_major << "." << cusp_minor << std::endl;

	// H has storage for 4 integers
	thrust::host_vector<int> H(4);

	//initialize individual elements
	H[0] = 14;
	H[1] = 20;
	H[2] = 38;
	H[3] = 46;

	// H.size() returns the size of vector H
	std::cout << "H has size " << H.size() << std::endl;

	//print contents of H
	for(unsigned i = 0;i < H.size(); i++)
		std::cout << "H[" << i << "] = " << H[i] << std::endl;

	// resize H
	H.resize(2);

	std::cout << "After resize, H now has size " << H.size() << std::endl;

	//print contents of H
	for(unsigned i = 0;i < H.size(); i++)
		std::cout << "H[" << i << "] = " << H[i] << std::endl;

	// Copy host_vector H to device_vector D
	thrust::device_vector<int> D = H;

	// elements of D can be modfied
	D[0] = 99;
	D[1] = 88;
	
	// print contents of D
	for(unsigned i = 0;i < D.size(); i++)
		std::cout << "D[" << i << "] = " << D[i] << std::endl;

	// print contents of H
	for(unsigned i = 0;i < H.size(); i++)
		std::cout << "H[" << i << "] = " << H[i] << std::endl;
	
	// H and D are automatically deleted when the function returns
}

void thrustMULTIPLYExample()
{
	thrust::device_vector<cfloat> D1(100000, cfloat(1,2));
	thrust::device_vector<cfloat> D2(100000, cfloat(2,3));
	thrust::device_vector<cfloat> out(100000);

	thrust::transform(D1.begin(), D1.end(), D2.begin(), out.begin(), thrust::multiplies<cfloat>());

	// print contents
	for(unsigned i = 0;i < D1.size()/10000; i++){
		cfloat cc = D1[i];
		std::cout << "D1[" << i << "] = " << cc << std::endl;
	}

	// print contents
	for(unsigned i = 0;i < D2.size()/10000; i++){
		cfloat cc = D2[i];
		std::cout << "D2[" << i << "] = " << cc << std::endl;
	}

	// print contents
	for(unsigned i = 0;i < out.size()/10000; i++){
		cfloat cc = out[i];
		std::cout << "out[" << i << "] = " << cc << std::endl;
	}

}


/* host functions */
void quitProgramPrompt(bool success)
{
  int c;
  if(success)
	printf( "\nProgram Executed Successfully. Press ENTER to quit program...\n" );
  else
	printf( "\nProgram Execution Failed. Press ENTER to quit program...\n" );
  fflush( stdout );
  do c = getchar(); while ((c != '\n') && (c != EOF));
}